#include "hip/hip_runtime.h"
/**
 * gesummv.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "gesummv.cuh"
#include <polybench.h>
#include <polybenchUtilFuncts.h>

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

#define GPU_DEVICE 0

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 43532.0f
#define BETA 12313.0f

#define RUN_ON_CPU


void gesummv(int nn, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A,NN,NN,nn,nn), DATA_TYPE POLYBENCH_2D(B,NN,NN,nn,nn), DATA_TYPE POLYBENCH_1D(tmp,NN,nn),
		DATA_TYPE POLYBENCH_1D(x,NN,nn), DATA_TYPE POLYBENCH_1D(y,NN,nn))
{
	int i, j;
	
	for (i = 0; i < _PB_NN; i++)
	{
		tmp[i] = 0;
		y[i] = 0;
		for (j = 0; j < _PB_NN; j++)
		{
			tmp[i] = A[i][j] * x[j] + tmp[i];
			y[i] = B[i][j] * x[j] + y[i];
		}
		
		y[i] = alpha * tmp[i] + beta * y[i];
	}
}


void init(int nn, DATA_TYPE *alpha, DATA_TYPE *beta, DATA_TYPE POLYBENCH_2D(A,NN,NN,nn,nn), DATA_TYPE POLYBENCH_2D(B,NN,NN,nn,nn),
	DATA_TYPE POLYBENCH_1D(x,NN,nn))
{
  	int i, j;

	*alpha = 43532;
	*beta = 12313;

 	for (i = 0; i < nn; i++)
    	{
    		x[i] = ((DATA_TYPE) i) / NN;
      	
		for (j = 0; j < nn; j++) 
		{
			A[i][j] = ((DATA_TYPE) i*j) / NN;
			B[i][j] = ((DATA_TYPE) i*j) / nn;
		}
    }
}


void compareResults(int nn, DATA_TYPE POLYBENCH_1D(y,NN,nn), DATA_TYPE POLYBENCH_1D(y_outputFromGpu,NN,nn))
{
	int i, fail;
	fail = 0;
	
	for (i=0; i<nn; i++) 
	{
		if (percentDiff(y[i], y_outputFromGpu[i]) > PERCENT_DIFF_ERROR_THRESHOLD) 
		{
			fail++;
		}
	}
	
	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}


void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gesummv_kernel(int nn, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A,NN,NN,nn,nn), DATA_TYPE POLYBENCH_2D(B,NN,NN,nn,nn),
	DATA_TYPE POLYBENCH_1D(tmp,NN,nn), DATA_TYPE POLYBENCH_1D(x,NN,nn), DATA_TYPE POLYBENCH_1D(y,NN,nn))
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < _PB_NN)
	{
		int j;
		for(j = 0; j < _PB_NN; j++)
		{	
			tmp[i] += A[i][j] * x[j];
			y[i] += B[i][j] * x[j];
		}
		y[i] = alpha * tmp[i] + beta  * y[i];
	}
}

void gesummvCuda(int nn, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A,NN,NN,nn,nn), DATA_TYPE POLYBENCH_2D(B,NN,NN,nn,nn),
		DATA_TYPE POLYBENCH_1D(tmp,NN,nn), DATA_TYPE POLYBENCH_1D(x,NN,nn), DATA_TYPE POLYBENCH_1D(y,NN,nn),  
		DATA_TYPE POLYBENCH_1D(y_outputFromGpu,NN,nn))
{
	DATA_TYPE (*A_gpu)[NN];
	DATA_TYPE (*B_gpu)[NN];
	DATA_TYPE *x_gpu;
	DATA_TYPE *y_gpu;
	DATA_TYPE *tmp_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NN * NN);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NN * NN);
	hipMalloc((void **)&x_gpu, sizeof(DATA_TYPE) * NN);
	hipMalloc((void **)&y_gpu, sizeof(DATA_TYPE) * NN);
	hipMalloc((void **)&tmp_gpu, sizeof(DATA_TYPE) * NN);
	
	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NN * NN, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NN * NN, hipMemcpyHostToDevice);
	hipMemcpy(x_gpu, x, sizeof(DATA_TYPE) * NN, hipMemcpyHostToDevice);
	hipMemcpy(y_gpu, y, sizeof(DATA_TYPE) * NN, hipMemcpyHostToDevice);
	hipMemcpy(tmp_gpu, tmp, sizeof(DATA_TYPE) * NN, hipMemcpyHostToDevice);

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((unsigned int)ceil( ((float)NN) / ((float)block.x) ), 1);


	/* Start timer. */
  	polybench_start_instruments;

	gesummv_kernel<<< grid, block>>>(nn, alpha, beta, A_gpu, B_gpu, tmp_gpu, x_gpu, y_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	printf("GPU Time in seconds:\n");
  	polybench_stop_instruments;
 	polybench_print_instruments;

	hipMemcpy(y_outputFromGpu, y_gpu, sizeof(DATA_TYPE) * NN, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(x_gpu);
	hipFree(y_gpu);
	hipFree(tmp_gpu);
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int nn,
		 DATA_TYPE POLYBENCH_1D(y,NN,nn))

{
  int i;

  for (i = 0; i < nn; i++) {
    fprintf (stderr, DATA_PRINTF_MODIFIER, y[i]);
    if (i % 20 == 0) fprintf (stderr, "\n");
  }
}


int main(int argc, char *argv[])
{
	/* Retrieve problem size. */
	int nn = NN;

	/* Variable declaration/allocation. */
	DATA_TYPE alpha;
	DATA_TYPE beta;
	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NN,NN,nn,nn);
	POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,NN,NN,nn,nn);
	POLYBENCH_1D_ARRAY_DECL(tmp,DATA_TYPE,NN,nn);
	POLYBENCH_1D_ARRAY_DECL(x,DATA_TYPE,NN,nn);
	POLYBENCH_1D_ARRAY_DECL(y,DATA_TYPE,NN,nn);
	POLYBENCH_1D_ARRAY_DECL(y_outputFromGpu,DATA_TYPE,NN,nn);

	init(nn, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(x));
	
	GPU_argv_init();
	gesummvCuda(nn, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(tmp), POLYBENCH_ARRAY(x), POLYBENCH_ARRAY(y),  
		POLYBENCH_ARRAY(y_outputFromGpu));
	
	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		gesummv(nn, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(tmp), POLYBENCH_ARRAY(x), POLYBENCH_ARRAY(y));
		
		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
	  	polybench_stop_instruments;
	 	polybench_print_instruments;
	
		compareResults(nn, POLYBENCH_ARRAY(y), POLYBENCH_ARRAY(y_outputFromGpu));

	#else //prevent dead code elimination

		polybench_prevent_dce(print_array(nn, POLYBENCH_ARRAY(y_outputFromGpu)));

	#endif //RUN_ON_CPU


	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(B);  
	POLYBENCH_FREE_ARRAY(tmp);
	POLYBENCH_FREE_ARRAY(x);  
	POLYBENCH_FREE_ARRAY(y);
	POLYBENCH_FREE_ARRAY(y_outputFromGpu);

	return 0;
}

#include <polybench.c>
